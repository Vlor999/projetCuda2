#include "hip/hip_runtime.h"
#include "prefixsum.h"
#include "helper/hip/hip_vector_types.h"
#include "helper/cuda_helper_host.h"

#define BLOCK 256 // 512 does not inprove anything 

PrefixSumBlending_GPU::PrefixSumBlending_GPU() 
{
}

PrefixSumBlending_GPU::~PrefixSumBlending_GPU() 
{
}

void PrefixSumBlending_GPU::setup(uint2 dimensions, uint32_t samples_per_pixel) {
    CUDA_SYNC_CHECK_THROW();
}

void PrefixSumBlending_GPU::finalize() 
{

}

__global__ void prefixAndColor(uint32_t n_pixels, uint32_t n_samples_per_pixel, const float *alpha, const float3 *colors, float3 *img_out) {
    uint32_t pos = blockIdx.x * blockDim.x + threadIdx.x;

    float3 sum = make_float3(0.0f, 0.0f, 0.0f);
    uint32_t base = pos * n_samples_per_pixel;

    float acc = 1.0f;
    for (uint32_t i = 0; i < n_samples_per_pixel; ++i) {
        uint32_t writtingPos = base + i;
        float ai = alpha[writtingPos];

        float3 ci = colors[writtingPos];
        sum += ci * ai * acc;
        acc *= (1.0f - ai);
    }

    img_out[pos] = sum;
}

void PrefixSumBlending_GPU::run(DatasetGPU &data, float3 *d_img_out)
{
    uint32_t grid = (data._n_pixels + BLOCK - 1) / BLOCK;
    prefixAndColor<<<grid, BLOCK>>>(data._n_pixels, data._samples_per_pixel, data._alphas, data._colors,d_img_out);

    CUDA_SYNC_CHECK_THROW();
}

